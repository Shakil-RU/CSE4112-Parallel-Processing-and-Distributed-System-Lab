#include "hip/hip_runtime.h"
//!nvcc -arch=sm_75 search_phonebook.cu -o search_phonebook
//!time ./search_phonebook SHAKIL 100 > output1.txt

%%writefile search_phonebook.cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

using namespace std;

struct Contact {
  char id[50];
  char name[50];
  char number[50];
};

__device__ bool check(char* str1, char* str2, int len) {
  for(int i = 0; str1[i] != '\0'; i++) {
    int j = 0;
    while(str1[i+j] != '\0' && str2[j] != '\0' && str1[i+j] == str2[j]) {
      j++;
    }
    if(j == len-1) {
      return true;
    }
  }
  return false;
}

__global__ void searchPhonebook(Contact* phonebook, int num_contacts, char* search_name, int name_length) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < num_contacts) {
    if(check(phonebook[idx].name, search_name, name_length)) {
      printf("%s %s %s\n", phonebook[idx].id, phonebook[idx].name, phonebook[idx].number);
    }
  }
}

int main(int argc, char* argv[]) {
  if(argc != 3) {
    cerr << "Usage: " << argv[0] << " <search_name> <num_threads>" << endl;
    return 1;
  }

  string search_name = argv[1];
  int num_threads = atoi(argv[2]);
  // Mount Google Drive and copy the location
  string file_name = "/content/drive/MyDrive/Dataset/labtest_dataset1.txt";
  //string file_name = "phonebook1.txt";

  vector<Contact> phonebook;

  ifstream file(file_name);
  if(!file.is_open()) {
    cerr << "Error opening file: " << file_name << endl;
    return 1;
  }
  else {
    Contact contact;
    string line;
    while(getline(file, line)) {
      /* Format: "id","name","phone_number"
      int pos = line.find(",");
      strcpy(contact.id, line.substr(1, pos-2).c_str());
      line = line.substr(pos+1);
      pos = line.find(",");
      strcpy(contact.name, line.substr(1, pos-2).c_str());
      strcpy(contact.number, line.substr(pos+2, line.size()-pos-4).c_str());
      phonebook.push_back(contact);
      */

      // Format: "name","phone_number"
      int pos = line.find(",");
      // Extract name (without the quotes)
      strcpy(contact.name, line.substr(1, pos - 2).c_str());

      // Extract number (also without quotes)
      strcpy(contact.number, line.substr(pos + 2, line.size() - pos - 4).c_str());

      phonebook.push_back(contact);
    }
    file.close();
  }
  int num_contacts = phonebook.size();
  Contact* device_phonebook;
  hipMalloc((void**)&device_phonebook, sizeof(Contact)*num_contacts);
  hipMemcpy(device_phonebook, phonebook.data(), sizeof(Contact)*num_contacts, hipMemcpyHostToDevice);

  int name_length = search_name.length() + 1;
  char* device_search_name;
  hipMalloc((void**)&device_search_name, name_length);
  hipMemcpy(device_search_name, search_name.c_str(), name_length, hipMemcpyHostToDevice);

  for(int i = 0; i < num_contacts; i += num_threads) {
    int thread_count = min(num_contacts-i, num_threads);
    searchPhonebook<<<1, thread_count>>>(device_phonebook + i, thread_count, device_search_name, name_length);
    hipDeviceSynchronize();
  }

  hipFree(device_phonebook);
  hipFree(device_search_name);

  return 0;
}
